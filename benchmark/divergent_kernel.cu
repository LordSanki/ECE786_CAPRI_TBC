
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <string.h>

#define DATA_LENGTH 100

#define CUDA_CALL(X) X; // {if(cudaError == X){printf("Error Calling %s at line %s\n", #X, __LINE__);}}
float * genInput(int l);

void verify(float *a, float *b, float *c, int l);

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  int i=0;
  int tid = threadIdx.x;

  for(i=0; i<DATA_LENGTH; i++){
    // first half of first warp and second half of 2nd warp
    if(tid < 16 || tid > 47)
      out[i] = in1[i]+in2[i];
    else
      out[i] = in1[i]+in2[i];
  }

  for(i=0; i<DATA_LENGTH; i++){
    // only even threads not compactable
    if(threadIdx.x%2 == 0 )
      out[i] = in1[i]+in2[i];
    else
      out[i] = in1[i]+in2[i];
  }
  out[0] = in1[0] + in2[0];
}

int main(int argc, char **argv) {
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;
  struct timeval t;
  gettimeofday(&t, NULL);
  srand(t.tv_sec);

  inputLength = DATA_LENGTH;

  hostInput1 = genInput(inputLength);
  hostInput2 = genInput(inputLength);
  hostOutput = ( float * )malloc(inputLength * sizeof(float));

  //@@ Allocate GPU memory here
  CUDA_CALL(hipMalloc((void**)&deviceInput1, inputLength*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&deviceInput2, inputLength*sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&deviceOutput, inputLength*sizeof(float)));

  //@@ Copy memory to the GPU here
  CUDA_CALL(hipMemcpy(deviceInput1, hostInput1, sizeof(float)*inputLength, hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(deviceInput2, hostInput2, sizeof(float)*inputLength, hipMemcpyHostToDevice));

  //@@ Initialize the grid and block dimensions here
  dim3 numBlocks(1,1,1);
  //dim3 numThreads(ThreadsPerBlock,1,1);
  dim3 numThreads(64,1,1);

  //@@ Launch the GPU Kernel here
  vecAdd<<<numBlocks, numThreads>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CALL(hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(float), hipMemcpyDeviceToHost));

  //@@ Free the GPU memory here
  CUDA_CALL(hipFree(deviceInput1));
  CUDA_CALL(hipFree(deviceInput2));
  CUDA_CALL(hipFree(deviceOutput));

  verify(hostInput1, hostInput2, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}

float * genInput(int l)
{
  int i;
  float * arr = (float*)malloc(l*sizeof(float));
  for(i=0; i<l; i++){
    arr[i] = rand();
    arr[i] = arr[i]/rand();
  }
  return arr;
}

void verify(float *a, float *b, float *c, int l)
{
  char buff1[50] = {0};
  char buff2[50] = {0};
  int i;
  for(i=0; i<l; i++){
    float d = a[i]+b[i];
    sprintf(buff1, "%1.8f", d); 
    sprintf(buff2, "%1.8f", c[i]);
    if(strcmp(buff1, buff2) != 0){
      printf("ERROR at index %d, Exp %1.8f Got %1.8f\n",i,d,c[i]);
      break;
    }
  }
}


